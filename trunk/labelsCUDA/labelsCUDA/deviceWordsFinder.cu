#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
//#include "CudaUtil.h"

#include <iostream>
//#include <FileLoader.h>
#include "FileStruct.h"
#include "Word.h"

#include "..\WordFinder\WordFinderLib.h"
#include "..\cudpp\include\cudpp.h"

#include "Buffer.h"
#include "deviceWordsFinder.h"


__global__ void
device_MarkAllWords(char* text, int len, int* terminatedSymbols)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
	int ty = threadIdx.y;

	extern __shared__ int sData[];
	
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int r;
	char c = 0; 
	
	if (idx < len-1)
	{
		c = text[idx];
	}
	r = ((
		(c == ' ')||
		(c == '.')||
		(c == ',')||
		(c == '!')||
		(c == '?')) 
		|| (idx==0));	
	sData[tx] = r;
	

	__syncthreads();

	if (idx < len-1)
	{
		if (tx!=0)
		{
			int r0 = sData[tx-1];
			r = (r0) && (!r);
		}
		else
			if (idx!=0)
			{
				c = text[idx-1];
				int rprev = (
					(c == ' ')||
					(c == '.')||
					(c == ',')||
					(c == '!')||
					(c == '?'));	
				r = (rprev) && (!r);
			}
		terminatedSymbols[idx] = r;
	}	

}

__device__ Transition* table;

__global__ void
device_FindAllWords( Transition* table, char* text, int len, int* position, size_t* count, int* words)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
	int ty = threadIdx.y;

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < *count)
	{
		int state = 0;
		int pos = position[idx];
		int output;
		//pos++;
		Transition trans;
		do
		{
			trans = GetTransaction(table, state, text[pos]);
			pos++;
			state = trans.NextState;
		}
		while((state != 0) && (pos < len));
		words[idx]	= trans.Output;			
	}
}

__global__ void
device_NormalizeAllWords( unsigned int* words, size_t count)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
	int ty = threadIdx.y;

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < count)
	{
		unsigned int r = (words[idx])?1:0;
		words[idx] = r;
	}
}

CUDPPHandle scanplan1 = 0;
CUDPPHandle scanplan2 = 0;
int mem_size2 = sizeof( int) * 10240 * 1024;
Buffer terminatedSymbolsBuf(mem_size2);
Buffer d_odataBuf(mem_size2);
Buffer pwordsCountBuf(sizeof(size_t)*2);

Buffer wordsId(mem_size2);
Buffer valid(mem_size2);
Buffer keyWordsId(mem_size2);

void deviceFindAllWordsPrepare(TransitionsTable * transTable, size_t len)
{
	hipMalloc((void**)&table, transTable->FullSize);
	
	hipMemcpy(table, transTable->Table, transTable->FullSize, hipMemcpyHostToDevice);

	CUDPPConfiguration config;	
	config.datatype = CUDPP_INT;
	config.algorithm = CUDPP_COMPACT;
	config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE |CUDPP_OPTION_INDEX;
    
    CUDPPResult result = cudppPlan(&scanplan1, config, len, 1, 0); 

	CUDPPConfiguration config2;	

	config2.datatype = CUDPP_INT;
	config2.algorithm = CUDPP_COMPACT;
	config2.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE ;	

	result = cudppPlan(&scanplan2, config2, len, 1, 0); 
	
}

void deviceFindAllWords( char* text, int len, Word* words, int* count, int * allWords, int* allCount)
{
	// setup execution parameters
	int threadsNum = 128;
    dim3 threads(threadsNum, 1);
    dim3 grid((len-1)/threadsNum+1,1);
	int* terminatedSymbols;
	int num_elements = len;
	int mem_size = sizeof( int) * num_elements;
	int sharedMemSize = threadsNum * sizeof(int);
//	Buffer terminatedSymbolsBuf(mem_size);
	terminatedSymbols = (int*)terminatedSymbolsBuf.GetDevice();
	
	device_MarkAllWords<<< grid, threads, sharedMemSize >>>(text, len, terminatedSymbols);
	

	// allocate device memory output arrays
	//Buffer d_odataBuf(mem_size);
	int* d_odata = (int*)d_odataBuf.GetDevice();
	
	//Buffer pwordsCountBuf(sizeof(size_t));
	size_t* pwordsCount = (size_t*)pwordsCountBuf.GetDevice();

	cudppCompact(scanplan1, d_odata, pwordsCount, text,(unsigned int*) terminatedSymbols, len);
	


	//Buffer wordsId(pwordsCount, sizeof(int));
	//Buffer valid(pwordsCount, sizeof(int));
	//Buffer keyWordsId(pwordsCount, sizeof(int));

		
	device_FindAllWords<<< grid, threads, sharedMemSize >>>(table, text, len, d_odata, pwordsCount, (int*) wordsId.GetDevice() );
		
	unsigned int* w = (unsigned int*) wordsId.GetDevice();
	int countWords;
	hipMemcpy( &countWords, pwordsCount, sizeof(int), hipMemcpyDeviceToHost);

	hipMemcpy( valid.GetDevice(), wordsId.GetDevice(), countWords*sizeof(int), hipMemcpyDeviceToDevice);
	device_NormalizeAllWords<<< grid, threads >>>((unsigned int *)valid.GetDevice(), countWords);
	
	
	cudppCompact(scanplan2,( void*) keyWordsId.GetDevice(), pwordsCount, wordsId.GetDevice(), (unsigned int *)valid.GetDevice(), (size_t)(countWords));	
	
	hipMemcpy(count, pwordsCount, sizeof(int), hipMemcpyDeviceToDevice);
	
	//hipFree(pwordsCount);

}

