#include "hip/hip_runtime.h"
// test2.cpp : Defines the entry point for the console application.
//

//#include "stdafx.h"
//
//
//int _tmain(int argc, _TCHAR* argv[])
//{
//	return 0;
//}

//#include <C:\Program Files\NVIDIA Nexus 1.0\CUDA Toolkit\v3.0\Win32\CUDA\include\thrust/version.h>
//#include <C:\Program Files\NVIDIA Nexus 1.0\CUDA Toolkit\v3.0\Win32\CUDA\include\thrust/device_vector.h>

#include <hip/hip_runtime_api.h>
#include "CudaUtil.h"
//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>
//
//#include <thrust/copy.h>
//#include <thrust/fill.h>
//#include <thrust/sequence.h>

#include <iostream>
#include <time.h>
#include <vector>;
#include "FileStruct.h"
#include "Word.h"

#include "..\WordFinder\WordFinderLib.h"
#include "..\cudpp\include\cudpp.h"
#include "deviceWordsFinder.h"



void Foo();

__global__ void
test(char *a, int len)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
	int ty = threadIdx.y;

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < len)
	{
		a[idx] = a[idx]+1;
	}
}

int PrintDevices(int deviceCount, int deviceSelected)
{
    hipError_t err = hipSuccess;

    hipDeviceProp_t deviceProperty;
    for (int currentDeviceId = 0; currentDeviceId < deviceCount; ++currentDeviceId)
    {
        memset(&deviceProperty, 0, sizeof(hipDeviceProp_t));
        err = hipGetDeviceProperties(&deviceProperty, currentDeviceId);
        //CheckConditionXR_(err == hipSuccess, err);

        printf("\ndevice name: %s", deviceProperty.name);
        if (currentDeviceId == deviceSelected)
        {
            printf("    <----- creating hipCtx_t on this");    
        }
        printf("\n");

        printf("device sharedMemPerBlock: %d \n", deviceProperty.sharedMemPerBlock);
        printf("device totalGlobalMem: %d \n", deviceProperty.totalGlobalMem);
        printf("device regsPerBlock: %d \n", deviceProperty.regsPerBlock);
        printf("device warpSize: %d \n", deviceProperty.warpSize);
        printf("device memPitch: %d \n", deviceProperty.memPitch);
        printf("device maxThreadsPerBlock: %d \n", deviceProperty.maxThreadsPerBlock);
        printf("device maxThreadsDim[0]: %d \n", deviceProperty.maxThreadsDim[0]);
        printf("device maxThreadsDim[1]: %d \n", deviceProperty.maxThreadsDim[1]);
        printf("device maxThreadsDim[2]: %d \n", deviceProperty.maxThreadsDim[2]);
        printf("device maxGridSize[0]: %d \n", deviceProperty.maxGridSize[0]);
        printf("device maxGridSize[1]: %d \n", deviceProperty.maxGridSize[1]);
        printf("device maxGridSize[2]: %d \n", deviceProperty.maxGridSize[2]);
        printf("device totalConstMem: %d \n", deviceProperty.totalConstMem);
        printf("device major: %d \n", deviceProperty.major);
        printf("device minor: %d \n", deviceProperty.minor);
        printf("device clockRate: %d \n", deviceProperty.clockRate);
        printf("device textureAlignment: %d \n", deviceProperty.textureAlignment);
        printf("device deviceOverlap: %d \n", deviceProperty.deviceOverlap);
        printf("device multiProcessorCount: %d \n", deviceProperty.multiProcessorCount);

        printf("\n");
    }

    return hipSuccess;
}

int main()
{
	//PrintDevices(1,0);

	Foo();
	return;


	//char * buf;
	//int size = LoadFile(".\\goog0.txt", buf);
	//if (size < 10)
	//{
	//	std::cout << "error opening file";
	//	return;
	//}
	//printf("size: %d, text: %s \n", size, buf);

	//int len = 320;
	////// allocate device memory
 ////   int* a;
 ////   hipMalloc((void**) &a, len * sizeof (int));
 ////   int* b;
 ////   hipMalloc((void**) &b, len * sizeof (int));

	//char* deviceBuf;
 //   hipMalloc((void**) &deviceBuf, size);
	//hipMemcpy(deviceBuf, buf, size, hipMemcpyHostToDevice);
	FileStruct* file = new FileStruct(".\\goog0.txt");
	size_t size = file->GetSize();
	
	char * deviceBuffer = file->GetDeviceBuffer();
	// setup execution parameters
    dim3 threads(512, 1);
    dim3 grid(size/512,1);

    // execute the kernel
    test<<< grid, threads >>>(deviceBuffer, size);
	
	char* buf =(char*) malloc(size + 1);
	hipMemcpy(buf, deviceBuffer, size, hipMemcpyDeviceToHost);

	//hipFree(deviceBuf);
    // print a
	printf("text2: %s", buf);
    //for(int i = 0; i < size; i++)
    //    std::cout << "A[" << i << "] = " << buf[i] << std::endl;
	free(buf);
	getchar();
	delete file;
    return 0;
}

char* Test2(char* text, size_t size)
{
	char * a;
    hipMalloc((void**) &a, size);

	hipMemcpy(a, text, size, hipMemcpyHostToDevice);
	
	return a;
}


int host_FindAllWords(Transition* table, char* text, Word* words )
{
	std::vector<int> w;
	for (int i = 0; text[i] != 0; ++i)
	{
		char c = text[i];
		bool r = ((
			(c == ' ')||
			(c == '.')||
			(c == ',')||
			(c == '!')||
			(c == '?')) 
			|| (i==0));	
		if (r)
		{
			w.push_back(i);
		}
	}

	int wordsCount = 0;
	for (int k=0; k < w.size()-1; k++)
	{
		int state = 0;
		Transition trans;

		for (int i = (w[k]); i <  (w[k+1]); ++i)
		{
			trans = GetTransaction(table, state, text[i]);

			if (trans.Output != 0)
			{
				Word word;
				word.Id = trans.Output;
				word.Pos = i;
				words[wordsCount++] = word;
			}
			state = trans.NextState;

		}
	}
	return wordsCount;
} 

__global__ void
device_WatchDebug(char * str)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
	int ty = threadIdx.y;

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	char * s = str;
	s[tx] = 'B';
}

bool FindedWordsEqual(Word * w1, int count1, Word* w2, int count2)
{
	bool result;
	if (count1 = count2)
	{
		result = (memcmp(w1, w2, count1* sizeof(Word)) == 0);
	}
	else
	{
		result = false;
	}
	return result;
}

void Foo()
{
	WordFinder* finder = CreateWordFinder();
	FILE* f = fopen(".\\words.txt","rt");
	char* tmpBuf = new char[64];
	std::vector<std::string> words;
	while (!feof(f))
	{
		fgets(tmpBuf, 64, f);
		std::string word = tmpBuf;
		words.push_back(word);	
	}
	finder->AddWords( words );
	delete[] tmpBuf;
	TransitionsTable* table = finder->Generate();
	
	FileStruct* file = new FileStruct(".\\goog0b.txt");
	char* text = file->GetHostBuffer();
	
	event_pair time;
	start_timer(&time);
	Word* findedWords = new Word[file->GetSize()];
	int host_count;

	clock_t start, end;
	
	start = clock();

	for (int i = 0; i < 1000; i++)
	{
		host_count = host_FindAllWords(table->Table , text, findedWords);
	}

	end = clock();

	double host_time= difftime(end, start)/CLOCKS_PER_SEC*1000;
	printf("%s took %.1f ms\n","CPU", host_time);
	//float host_time = (end-init)/CLOCKS_PER_SEC; //stop_timer(&time, "CPU word finder");
	

	size_t size = file->GetSize();

	// setup execution parameters
    dim3 threads(512, 1);
    dim3 grid(size/512,1);
	

    // execute the kernel	
	//Transition* device_table = (Transition*)GetDeviceMemory(table->Table, table->Size);

	check_cuda_error("Host to device Mem cpy:");
	Buffer device_wordsCountBuf(sizeof(int) * 2);
	int* pDeviceCount = (int*)device_wordsCountBuf.GetDevice();  
	Buffer device_findedWordsBuf(512 * sizeof(Word));
	Word* device_findedWords = (Word*)device_findedWordsBuf.GetDevice();
	char* device_text = file->GetDeviceBuffer();
	Buffer allWords(sizeof(int)*size/4);
	Buffer allWordsCount(sizeof(int));

	deviceFindAllWordsPrepare(table, size);
	deviceFindAllWords( device_text, size, device_findedWords,  pDeviceCount,
			(int*)allWords.GetDevice(), (int*)allWordsCount.GetDevice());
	start = clock();
	start_timer(&time);
	for (int i = 0; i < 1000; i++)
	{
		deviceFindAllWords( device_text, size, device_findedWords,  pDeviceCount,
			(int*)allWords.GetDevice(), (int*)allWordsCount.GetDevice());
	}
	//device_FindAllWords<<< grid, threads >>>(device_table, device_text, size, device_findedWords,  pDeviceCount,
	//	allWords.GetDevice(), allWordsCount.GetDevice());
	end = clock();

	double device_time= difftime(end, start)/CLOCKS_PER_SEC*1000;
	printf("%s took %.1f ms\n","GPU", device_time);

	int device_count = *((int*)device_wordsCountBuf.GetHost());
	Word* devicefindedWords = (Word*)device_findedWordsBuf.GetHost();
	//hipFree(device_table);
	check_cuda_error("CUDA:");
	//check_launch("CUDA word finder");
	device_time = stop_timer(&time, "GPU word finder");
	
	Buffer result(512);
	
	sprintf((char*)result.GetHost(),"CPU version time: %f, Count: %d ; Device version time: %f, count %d; allWord count: %d ",
		host_time, host_count, device_time, device_count, allWordsCount.GetHost());
	printf((char*)result.GetHost());
	device_WatchDebug<<< 1, 1 >>>((char*)result.GetDevice());
	
	delete[] findedWords;
	delete file;
	delete table;
}

