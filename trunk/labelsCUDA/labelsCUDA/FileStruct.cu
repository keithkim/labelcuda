#include "FileStruct.h"
#include <iostream>
#include <stdio.h>


FileStruct::FileStruct(char* fileName)
{
	_deviceBuffer = NULL;
	_buffer = NULL;
	_size = 0;
	Name = fileName;
}

FileStruct::~FileStruct(void)
{
	if (_buffer != NULL)
	{
		free(_buffer);
	}

	if (_deviceBuffer != NULL)
	{
		hipFree(_deviceBuffer);
	}
}


size_t FileStruct::LoadFile(char* path, char* &buffer)
{
	FILE *file;
	
	size_t fileLen;
	
	//Open file
	file = fopen(path, "rt");
	if (!file)
	{
		fprintf(stderr, "Unable to open file %s", path);
		return 0;
	}
	
	//Get file length
	fseek(file, 0, SEEK_END);
	fileLen=ftell(file);
	fseek(file, 0, SEEK_SET);

	//Allocate memory
	buffer=(char *)malloc(fileLen+1);
	if (!buffer)
	{
		fprintf(stderr, "Memory error!");
                                fclose(file);
		return 0;
	}

	//Read file contents into buffer
	fread(buffer, fileLen, 1, file);
	fclose(file);

	return fileLen;
}

char * FileStruct::GetHostBuffer()
{
	if (_buffer == NULL)
	{
		_size = this->LoadFile(Name, _buffer);
	}
	return _buffer;
}

size_t FileStruct::GetSize()
{
	if (_buffer == NULL)
	{
		_size = this->LoadFile(Name, _buffer);
	}
	return _size;
}

char * FileStruct::GetDeviceBuffer()
{
	if (_deviceBuffer == NULL)
	{
		char* buf = this->GetHostBuffer();
		size_t size = this->GetSize();
		hipMalloc((void**) &_deviceBuffer, size);
		hipMemcpy(_deviceBuffer, buf, size, hipMemcpyHostToDevice);
	}
	return _deviceBuffer;
}