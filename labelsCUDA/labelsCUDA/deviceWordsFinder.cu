#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
//#include "CudaUtil.h"

#include <iostream>
//#include <FileLoader.h>
#include "FileStruct.h"
#include "Word.h"

#include "..\WordFinder\WordFinderLib.h"
#include "..\cudpp\include\cudpp.h"

#include "Buffer.h"
#include "deviceWordsFinder.h"


__global__ void
device_MarkAllWords(char* text, int len, int* terminatedSymbols)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
	int ty = threadIdx.y;

	extern __shared__ int sData[];
	
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int r;
	char c = 0; 
	
	if (idx < len-1)
	{
		c = text[idx];
	}
	r = ((
		(c == ' ')||
		(c == '.')||
		(c == ',')||
		(c == '!')||
		(c == '?')) 
		|| (idx==0));	
	sData[tx] = r;
	

	__syncthreads();

	if (idx < len-1)
	{
		if (tx!=0)
		{
			int r0 = sData[tx-1];
			r = (r0) && (!r);
		}
		else
			if (idx!=0)
			{
				c = text[idx-1];
				int rprev = (
					(c == ' ')||
					(c == '.')||
					(c == ',')||
					(c == '!')||
					(c == '?'));	
				r = (rprev) && (!r);
			}
		terminatedSymbols[idx] = r;
	}	

}

__device__ Transition* table;

__global__ void
device_FindAllWords( Transition* table, char* text, int len, int* position, size_t* count, int* words)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
	int ty = threadIdx.y;

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < *count)
	{
		int state = 0;
		int pos = position[idx];
		int output;
		//pos++;
		Transition trans;
		do
		{
			trans = GetTransaction(table, state, text[pos]);
			pos++;
			state = trans.NextState;
		}
		while((state != 0) && (pos < len));
		words[idx]	= trans.Output;			
	}
}

__global__ void
device_NormalizeAllWords( unsigned int* words, size_t count)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
	int ty = threadIdx.y;

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < count)
	{
		unsigned int r = (words[idx])?1:0;
		words[idx] = r;
	}
}

void deviceFindAllWordsPrepare(TransitionsTable * transTable)
{
	hipMalloc((void**)&table, transTable->FullSize);
	
	hipMemcpy(table, transTable->Table, transTable->FullSize, hipMemcpyHostToDevice);
	
}

void deviceFindAllWords( char* text, int len, Word* words, int* count, int * allWords, int* allCount)
{
	// setup execution parameters
	int threadsNum = 512;
    dim3 threads(threadsNum, 1);
    dim3 grid((len-1)/threadsNum+1,1);
	int* terminatedSymbols;
	int num_elements = len;
	int mem_size = sizeof( int) * num_elements;
	int sharedMemSize = threadsNum * sizeof(int);

	hipMalloc(&terminatedSymbols, mem_size);
	device_MarkAllWords<<< grid, threads, sharedMemSize >>>(text, len, terminatedSymbols);
	

	// allocate device memory output arrays
    int* d_odata = NULL;

    hipMalloc( (void**) &d_odata, mem_size);

	CUDPPConfiguration config;	
	config.datatype = CUDPP_INT;
	config.algorithm = CUDPP_COMPACT;
	config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE |CUDPP_OPTION_INDEX;
    
    CUDPPHandle scanplan = 0;
    CUDPPResult result = cudppPlan(&scanplan, config, len, 1, 0); 
	//Buffer wordsCountBuf(sizeof(size_t));
	size_t* pwordsCount;
    hipMalloc( (void**) &pwordsCount, sizeof(size_t));

	cudppCompact(scanplan, d_odata, pwordsCount, text,(unsigned int*) terminatedSymbols, len);
	
	//printf("Words count: %d \n", *(int*)(wordsCountBuf.GetHost()) );
	//device_WatchDebug<<< 1, 1 >>>((char*)wordsCountBuf.GetDevice());

	Buffer wordsId(pwordsCount, sizeof(int));
	Buffer valid(pwordsCount, sizeof(int));
	Buffer keyWordsId(pwordsCount, sizeof(int));
	cudppDestroyPlan(scanplan);
		
	device_FindAllWords<<< grid, threads, sharedMemSize >>>(table, text, len, d_odata, pwordsCount, (int*) wordsId.GetDevice() );
	
	scanplan = 0;
	config.datatype = CUDPP_INT;
	config.algorithm = CUDPP_COMPACT;
	config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE ;	
	int countWords;
	hipMemcpy( &countWords, pwordsCount, sizeof(int), hipMemcpyDeviceToHost);
	result = cudppPlan(&scanplan, config, (size_t)(countWords), 1, 0); 
	unsigned int* w = (unsigned int*) wordsId.GetDevice();
	
	hipMemcpy( valid.GetDevice(), wordsId.GetDevice(), countWords*sizeof(int), hipMemcpyDeviceToDevice);
	device_NormalizeAllWords<<< grid, threads >>>((unsigned int *)valid.GetDevice(), countWords);

	cudppCompact(scanplan,( void*) keyWordsId.GetDevice(), pwordsCount, wordsId.GetDevice(), (unsigned int *)valid.GetDevice(), (size_t)(countWords));
	cudppDestroyPlan(scanplan);
	
	hipFree(d_odata);
	hipMemcpy(count, pwordsCount, sizeof(int), hipMemcpyDeviceToDevice);
	
	//hipFree(pwordsCount);

}

