#include "Buffer.h"
#include "stdio.h"
#include <hip/hip_runtime_api.h>

Buffer::Buffer(size_t size)
{
	_hostMemory = NULL;
	_deviceMemory = NULL;
	_size = size;
}
Buffer::Buffer(size_t* psize)
{
	_hostMemory = NULL;
	_deviceMemory = NULL;
	hipMemcpy(&_size, psize, sizeof(size_t), hipMemcpyDeviceToHost);
}

Buffer::Buffer(size_t* pcount, size_t elemSize)
{
	_hostMemory = NULL;
	_deviceMemory = NULL;
	hipMemcpy(&_size, pcount, sizeof(size_t), hipMemcpyDeviceToHost);
	_size *= elemSize;
}

Buffer::~Buffer(void)
{
	if (_hostMemory != NULL)
	{
		delete[](_hostMemory);
	}
	if (_deviceMemory != NULL)
	{
		hipFree(_deviceMemory);
	}
}

void* Buffer::GetHost()
{
	if (_hostMemory == NULL)
	{
		_hostMemory = new char[_size];
	}
	if (_deviceMemory != NULL)
	{
		hipMemcpy(_hostMemory, _deviceMemory, _size, hipMemcpyDeviceToHost);
	}
	return _hostMemory;
}

void* Buffer::GetDevice()
{
	if (_deviceMemory == NULL)
	{
		hipMalloc(&_deviceMemory, _size);
	}
	if (_hostMemory != NULL)
	{
		hipMemcpy(_deviceMemory,_hostMemory, _size, hipMemcpyHostToDevice);
	}
	return _deviceMemory;
}

inline size_t Buffer::GetSize()
{
	return _size;
}
