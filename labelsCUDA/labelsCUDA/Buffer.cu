#include "Buffer.h"
#include "stdio.h"
#include <hip/hip_runtime_api.h>

Buffer::Buffer(size_t size)
{
	_hostMemory = NULL;
	_deviceMemory = NULL;
	_size = size;
}

Buffer::~Buffer(void)
{
	if (_hostMemory != NULL)
	{
		free(_hostMemory);
	}
	if (_deviceMemory != NULL)
	{
		hipFree(_deviceMemory);
	}
}

void* Buffer::GetHost()
{
	if (_hostMemory == NULL)
	{
		_hostMemory = malloc(_size);
	}
	if (_deviceMemory != NULL)
	{
		hipMemcpy(_hostMemory, _deviceMemory, _size, hipMemcpyDeviceToHost);
	}
	return _hostMemory;
}

void* Buffer::GetDevice()
{
	if (_deviceMemory == NULL)
	{
		hipMalloc(&_deviceMemory, _size);
	}
	if (_hostMemory != NULL)
	{
		hipMemcpy(_deviceMemory,_hostMemory, _size, hipMemcpyHostToDevice);
	}
	return _deviceMemory;
}

inline size_t Buffer::GetSize()
{
	return _size;
}
