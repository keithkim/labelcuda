#include "hip/hip_runtime.h"
// test2.cpp : Defines the entry point for the console application.
//

//#include "stdafx.h"
//
//
//int _tmain(int argc, _TCHAR* argv[])
//{
//	return 0;
//}

//#include <C:\Program Files\NVIDIA Nexus 1.0\CUDA Toolkit\v3.0\Win32\CUDA\include\thrust/version.h>
//#include <C:\Program Files\NVIDIA Nexus 1.0\CUDA Toolkit\v3.0\Win32\CUDA\include\thrust/device_vector.h>

#include <hip/hip_runtime_api.h>

//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>
//
//#include <thrust/copy.h>
//#include <thrust/fill.h>
//#include <thrust/sequence.h>

#include <iostream>
//#include <FileLoader.h>
#include "FileStruct.h"

__global__ void
test(char *a, int len)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
	int ty = threadIdx.y;

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < len)
	{
		a[idx] = a[idx]+1;
	}
}

int PrintDevices(int deviceCount, int deviceSelected)
{
    hipError_t err = hipSuccess;

    hipDeviceProp_t deviceProperty;
    for (int currentDeviceId = 0; currentDeviceId < deviceCount; ++currentDeviceId)
    {
        memset(&deviceProperty, 0, sizeof(hipDeviceProp_t));
        err = hipGetDeviceProperties(&deviceProperty, currentDeviceId);
        //CheckConditionXR_(err == hipSuccess, err);

        printf("\ndevice name: %s", deviceProperty.name);
        if (currentDeviceId == deviceSelected)
        {
            printf("    <----- creating hipCtx_t on this");    
        }
        printf("\n");

        printf("device sharedMemPerBlock: %d \n", deviceProperty.sharedMemPerBlock);
        printf("device totalGlobalMem: %d \n", deviceProperty.totalGlobalMem);
        printf("device regsPerBlock: %d \n", deviceProperty.regsPerBlock);
        printf("device warpSize: %d \n", deviceProperty.warpSize);
        printf("device memPitch: %d \n", deviceProperty.memPitch);
        printf("device maxThreadsPerBlock: %d \n", deviceProperty.maxThreadsPerBlock);
        printf("device maxThreadsDim[0]: %d \n", deviceProperty.maxThreadsDim[0]);
        printf("device maxThreadsDim[1]: %d \n", deviceProperty.maxThreadsDim[1]);
        printf("device maxThreadsDim[2]: %d \n", deviceProperty.maxThreadsDim[2]);
        printf("device maxGridSize[0]: %d \n", deviceProperty.maxGridSize[0]);
        printf("device maxGridSize[1]: %d \n", deviceProperty.maxGridSize[1]);
        printf("device maxGridSize[2]: %d \n", deviceProperty.maxGridSize[2]);
        printf("device totalConstMem: %d \n", deviceProperty.totalConstMem);
        printf("device major: %d \n", deviceProperty.major);
        printf("device minor: %d \n", deviceProperty.minor);
        printf("device clockRate: %d \n", deviceProperty.clockRate);
        printf("device textureAlignment: %d \n", deviceProperty.textureAlignment);
        printf("device deviceOverlap: %d \n", deviceProperty.deviceOverlap);
        printf("device multiProcessorCount: %d \n", deviceProperty.multiProcessorCount);

        printf("\n");
    }

    return hipSuccess;
}

int main(void)
{
	PrintDevices(1,0);



	//char * buf;
	//int size = LoadFile(".\\goog0.txt", buf);
	//if (size < 10)
	//{
	//	std::cout << "error opening file";
	//	return;
	//}
	//printf("size: %d, text: %s \n", size, buf);

	//int len = 320;
	////// allocate device memory
 ////   int* a;
 ////   hipMalloc((void**) &a, len * sizeof (int));
 ////   int* b;
 ////   hipMalloc((void**) &b, len * sizeof (int));

	//char* deviceBuf;
 //   hipMalloc((void**) &deviceBuf, size);
	//hipMemcpy(deviceBuf, buf, size, hipMemcpyHostToDevice);
	FileStruct* file = new FileStruct(".\\goog0.txt");
	size_t size = file->GetSize();
	
	char * deviceBuffer = file->GetDeviceBuffer();
	// setup execution parameters
    dim3 threads(512, 1);
    dim3 grid(size/512,1);

    // execute the kernel
    test<<< grid, threads >>>(deviceBuffer, size);
	
	char* buf =(char*) malloc(size + 1);
	hipMemcpy(buf, deviceBuffer, size, hipMemcpyDeviceToHost);

	//hipFree(deviceBuf);
    // print a
	printf("text2: %s", buf);
    //for(int i = 0; i < size; i++)
    //    std::cout << "A[" << i << "] = " << buf[i] << std::endl;
	free(buf);
	getchar();
	delete file;
    return 0;
}

char* Test2(char* text, size_t size)
{
	char * a;
    hipMalloc((void**) &a, size);

	hipMemcpy(a, text, size, hipMemcpyHostToDevice);
	
	return a;
}